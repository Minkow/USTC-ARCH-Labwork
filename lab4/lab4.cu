#include "hip/hip_runtime.h"
﻿#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32

#define WA (10 * BLOCK_SIZE) // Matrix A width
#define HA (10 * BLOCK_SIZE) // Matrix A height
#define WB (20 * BLOCK_SIZE) // Matrix B width
#define HB WA  // Matrix B height
#define WC WB  // Matrix C width 
#define HC HA  // Matrix C height

void MatInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

__device__  float * GetSubMatrix(float *matrix, int m, int index, int width)
{
	return  matrix+width*BLOCK_SIZE*index+BLOCK_SIZE*m;
}

void CpuMul(float* C, const float* A, const float* B, int hA, int wA, int wB)
{
    for (int i = 0; i < hA; ++i)
        for (int j = 0; j < wB; ++j)
		{
            double sum = 0;
            for (int k = 0; k < wA; ++k) 
			{
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }
            C[i * wB + j] = (float)sum;
        }
}

__global__ void GpuMul1( float* C, float* A, float* B, int wA, int wB)
{
    int bx = blockIdx.x;
    int tx = threadIdx.x;    

	int idx = bx * blockDim.x + tx;
	int row = idx / wB;
	int column = idx % wB;

	float sum = 0;

	for(int i = 0; i < wA; ++i)
	{
		sum += A[row * wA + i] * B[i * wB + column];
	}
	C[row * wB + column] = sum;
//	printf("%d %d %d %d %f\n",row * wB + column,idx,row,column,sum);
}

__global__ void GpuMul2( float* C, float* A, float* B, int wA, int wB)
{
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;     

    float sum = 0;

    for (int m= 0; m<wA/BLOCK_SIZE; m++) 
	{
		float *subA=GetSubMatrix(A, m, by, wA);

		float *subB=GetSubMatrix(B, bx, m, wB);

        As[ty][tx] = *(subA+ wA * ty + tx);
        Bs[ty][tx] = *(subB+ wB * ty + tx);

        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k)
            sum += As[ty][k] * Bs[k][tx];
        __syncthreads();
    }

	float *subC=GetSubMatrix(C, bx, by, wB);
    *(subC + wB * ty + tx)= sum;
}

int main()
{
    // allocate host memory for matrices A and B
    int size_A = WA * HA;
    int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);

    int size_B = WB * HB;
    int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);
	clock_t start, finish;
	double time[3];

    MatInit(h_A, size_A);
    MatInit(h_B, size_B);

    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);

    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) ;
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) ;

    // allocate device memory for result
    int size_C = WC * HC;
    int mem_size_C = sizeof(float) * size_C;
    float* d1_C;
    hipMalloc((void**) &d1_C, mem_size_C);
	float* d2_C;
    hipMalloc((void**) &d2_C, mem_size_C);

    // allocate host memory for the result
	float* h0_C = (float*) malloc(mem_size_C);
    float* h1_C = (float*) malloc(mem_size_C);
	float* h2_C = (float*) malloc(mem_size_C);
    
	start=clock();
	int threads1 = BLOCK_SIZE * BLOCK_SIZE;
	int grid1 = WC*HC/threads1;
    GpuMul1<<< grid1, threads1 >>>(d1_C, d_A, d_B, WA, WB);
	hipDeviceSynchronize();
	finish=clock();  
	time[1]=(double)(finish-start)/CLOCKS_PER_SEC;
    hipMemcpy(h1_C, d1_C, mem_size_C, hipMemcpyDeviceToHost);

	start=clock();
	dim3 threads2(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid2(WC / threads2.x, HC / threads2.y);
    GpuMul2<<< grid2, threads2 >>>(d2_C, d_A, d_B, WA, WB);
	hipDeviceSynchronize();
	finish=clock();  
	time[2]=(double)(finish-start)/CLOCKS_PER_SEC;
    hipMemcpy(h2_C, d2_C, mem_size_C, hipMemcpyDeviceToHost);

	start=clock();
    CpuMul(h0_C, h_A, h_B, HA, WA, WB);
	finish=clock();
	time[0]=(double)(finish-start)/CLOCKS_PER_SEC;

//	for(int i=0;i<WC*HC;i++)
//		printf("%f %f %f\n",h0_C[i],h1_C[i],h2_C[i]);

	printf("%f  %f  %f",time[0],time[1],time[2]);

    // clean up memory
    free(h_A);
    free(h_B);
    free(h0_C);
	free(h1_C);
	free(h2_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d1_C);
	hipFree(d2_C);
    hipDeviceReset();
}

 